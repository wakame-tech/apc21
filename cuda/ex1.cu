
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel() {
    printf("Hello, World!\n");
}

int main() {
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    // cudaDeviceReset();
    return 0;
}

